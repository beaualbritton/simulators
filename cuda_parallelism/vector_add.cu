/**
 * Beau Albritton
 * Vector addition CUDA program.
 * 
 * This program performs vector addition on the GPU
 * using nvidia's CUDA language. Initializes two vectors (float *)
 * x and y, such that 
 * x[i] is a uniformly distributed random number within (0,1)
 * y is the complement of x[i] or just y = 1-x[i].
 * 
 * Allocates memory on both host (cpu) and device (gpu), copying
 * input vectors from the cpu to gpu. Then, it performs
 * parallel vector addition using a CUDA kernel `vecadd`
 * 
 * GPU time is measured in MS and max absolute error (from expected)
 * is computed after the kernel finishes.
 */
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
#define N (256*256*256)
__global__
void vecadd(float* x, float* y, float* z, int n);

int main()
{
	//Vectors stored on the GPU (device)
	float* d_x, *d_y, *d_z;
	//Vectors stored on the CPU (host)
	float* h_x, *h_y, *h_z;
	int n = N;

	//using blocksize 256 for each thread block
	const int blockSize = 256;//256 threads for each thread block
	//Number of blocks per thread
	int numBlocks = (n + blockSize - 1)/blockSize;	

	//Calling malloc to allocate memory for CPU vectors. 
	//size of float (in bytes) multiplied by N (defined as 256^3)
	h_x = (float *) malloc(n*sizeof(float));
	h_y = (float *) malloc(n*sizeof(float));
	h_z = (float *) malloc(n*sizeof(float));

	//cudaMalloc allocates memory on the GPU instead.
	hipMalloc( (void **) &d_x, n*sizeof(float));
	hipMalloc( (void **) &d_y, n*sizeof(float));
	hipMalloc( (void **) &d_z, n*sizeof(float));

	//Setting each x[i] in the vector x to a random number within (0,1)
	for(int i = 0; i < n; ++i)
	{
		h_x[i] = drand48();
		//y is the complement 
		h_y[i] = 1.0f-h_x[i];

	}
	//Copying the results computed in the CPU to the GPU
	hipMemcpy(d_x, h_x, n*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_y, h_y, n*sizeof(float), hipMemcpyHostToDevice);
	
	//Creating events to measure GPU time. Recording starttime first
	hipEvent_t start,stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start,0);

	//Calling the kernel so everything runs in parallel
	vecadd<<<numBlocks,blockSize>>>(d_x,d_y,d_z,N);

	//Now recording finish time
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);

	//Then getting elapsed time from start and stop events
	float elapsedTime;
	hipEventElapsedTime(&elapsedTime, start, stop);

	//synchronizing host & device
	hipDeviceSynchronize();
	//copying calculated result of z[i] from GPU to the CPU (for error checking)
	hipMemcpy(h_z, d_z, n*sizeof(float), hipMemcpyDeviceToHost);
	//Then destroying events
	hipEventDestroy(start);
	hipEventDestroy(stop);	

	//Now checking for max absolute error
	float errorSum=0.0f;

	for(int i = 0; i < n; ++i)
	{
		errorSum += fabs(1-h_z[i]);		
	}
	//printing
	printf("GPU time elasped: %.4f ms\n", elapsedTime);
	printf("Max Errors: %f", errorSum);

	//Now freeing up all the memory allocated on both cpu & gpu

	free(h_x); free(h_y); free(h_z);
	hipFree(d_x); hipFree(d_y); hipFree(d_z);

	return 0;
}

__global__
void vecadd(float* x, float* y, float* z, int n)
{
	//Computes the current index, effectively z[i] since kernel
	//Runs in parallel. Pretty much asking "what block am I in?"
	//and "Within that block, which thread?"
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	//ensure no out of bounds
	if(tid < n)
	{
		//summing the two (should be 1 since y = 1-x[i])
		z[tid] = x[tid]+y[tid];
	}
}
