/**
 * Beau Albritton
 * Matrix-vector Product
 * 
 * This program performs matrix-vector multiplication
 * on the GPU using nvidia's CUDA language.
 * Computes the product y = M * x in parallel,
 * where M is an n x n matrix
 * (one dimensional pointer indexing with M[i*n] for the row)
 * and x is a n x 1 vector.
 * 
 * Each entry in the matrix M and the vector x are initialized to be random
 * numbers in (0,1). 
 * 
 * This program uses unified memory via cudaMallocManaged()
 * for both device and host memory allocation, which removes
 * the need to copy from device to host.
 * 
 * Each thread computes a single row of the resulting vector y.
 * 
 * Execution time is measured on both the device (GPU) and the host (CPU).
 * 
 * The relative error between the two is then calculated.
 */
#include <stdlib.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
#define N (5000)

//defining functions
__global__
void mxv(float* M, float* x, float* y, int n);
void cpu_mxv(float* M, float* x, float* y, int n);
float relative_error(float* u, float* v,int n);

int main()
{
	//Unified memory pointers
	float* M, *x, *y;
	float* h_y; //Cpu float* for relative error calculation later (don't  want to override *y as the two need to be compared)
	int n = N;
	//Using blocksize 256 for each thread block
	const int blockSize = 256;
	//Number of blocks per thread
	int numBlocks = (n+blockSize-1)/blockSize;

	//cudaMalloc now manages memory for both host and device
	hipMallocManaged(&x,n*sizeof(float));
	hipMallocManaged(&y, n*sizeof(float));
	hipMallocManaged(&M,n*n*sizeof(float));
	//Allocating memory for cpu vector
	h_y = (float *) malloc(n*sizeof(float));

	//Initializing each item in the vector x to a uniformly distributed random number in (0,1)
	for(int i =0; i < n; ++i)
	{
		x[i] = drand48();
		for(int j = 0; j < n; ++j)
		{
			//same for matrix M
			M[i*n+j] = drand48();
		}

	}
	//Creating events to measure GPU time. Recording starttime first
	hipEvent_t start,stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start,0);
	//calling kernel for gpu
	mxv<<<numBlocks,blockSize>>>(M,x,y,n);
	//Recording finish time
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);

	//Calculating elapsed time on the GPU
	float gpuElapsedTime;
	hipEventElapsedTime(&gpuElapsedTime, start, stop);
	//Synchronize device and host
	hipDeviceSynchronize();
	hipEventDestroy(start);
	hipEventDestroy(stop);

	//Now calling same algorithm on the CPU
	//and measuring it
	struct timeval t1, t2;
	float cpuElapsedTime;
	//startTime
	gettimeofday(&t1, NULL);

	//doing some work (calling cpu MXV)

	cpu_mxv(M,x,h_y,n);
	//finishTime
	gettimeofday(&t2, NULL);

	//Simple elapsed time calculations (to ms)
	cpuElapsedTime = (t2.tv_sec - t1.tv_sec) * 1000.0;
	cpuElapsedTime += (t2.tv_usec - t1.tv_usec) / 1000;

	//Calculating relative error
	float relError = relative_error(y,h_y,n);

	//Then printing
	printf("cpu time %f ms\n", cpuElapsedTime);
	printf("gpu time %f ms\n", gpuElapsedTime);
	printf("relative error %.3f", relError);

	//Free ma bois
	hipFree(M);
	hipFree(x);
	hipFree(y);

	return 0;
}

__global__
void mxv(float* M, float* x, float* y, int n)
{
	float sum = 0.0;
	//Computes the current row. Pretty much asking "what block am I in?"
	//and "Within that block, which thread?"
	int row = threadIdx.x + blockDim.x *blockIdx.x;

	if(row < n)
	{
		//then for whatever row, multiply that by n (since M is an n x n vector and can only be represented by a 1D pointer float*)
		int idx = row * n;
		for(int col = 0; col < n; ++col)
		{
			sum = sum + M[idx + col] * x[col];
		}
		//y[i] is equal to the matrix-vector product
		y[row]=sum;
	}

}
void cpu_mxv(float* M, float* x, float* y, int n)
{
	//need outer foor loop for cpu since program isn't running in parallel.
	//otherwise exact same algorithm in mxv for gpu.
	for(int row = 0; row < n; ++row)
	{
		float sum = 0;
		for(int col = 0; col < n; ++col)
		{
			int idx = row * n + col;
			sum = sum+ M[idx] * x[col];
		}
		y[row] = sum;
	}
}
/**
 * Computes the relative error between two vectors U,V of size n.
 * Relative error given by || u - v || / || v ||
 * 
 * Simply put, the magnitude of the difference of the vectors U and V
 * divided by the overall magnitude of V.
 */
float relative_error(float* u, float* v,int n)
{
	//Sum for magnitude of the difference (for later square rooting)
	float magnitudeSum = 0.0f;
	//Sum for the mangitude of vec V
	float vSum = 0.0f;
	for(int i=0; i < n; ++i)
	{
		//Then square it and add it to sum according to the Euclidean norm (or just 2-norm)
		//https://en.wikipedia.org/wiki/Norm_(mathematics)#Euclidean_norm
		magnitudeSum += pow(u[i]-v[i], 2);
		vSum += pow(v[i],2);
	}
	//Finally square root 
	float magnitudeU_V = sqrtf(magnitudeSum);
	float magnitudeV = sqrtf(vSum);
	//Return the result 
	return magnitudeU_V / magnitudeV;
}
